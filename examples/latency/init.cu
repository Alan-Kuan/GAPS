#include "hip/hip_runtime.h"
#include "init.hpp"

#include <cstddef>

__global__ void __fillArray(int* arr, int tag) {
    size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
    arr[tid] = tag;
}

void init::fillArray(int* arr, size_t count, int tag) {
    dim3 grid_dim = 1;
    dim3 block_dim = count;
    if (count > 1024) {
        grid_dim = (count - 1) / 1024 + 1;
        block_dim = 1024;
    }
    __fillArray<<<grid_dim, block_dim>>>(arr, tag);
    hipDeviceSynchronize();
}