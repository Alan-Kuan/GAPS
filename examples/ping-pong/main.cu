#include "hip/hip_runtime.h"
#include <unistd.h>

#include <condition_variable>
#include <cstddef>
#include <cstdio>
#include <cstdlib>
#include <ctime>
#include <iostream>
#include <mutex>
#include <stdexcept>
#include <thread>

#include <hiprand/hiprand_kernel.h>
#include <iceoryx_posh/runtime/posh_runtime.hpp>

#include "node/publisher.hpp"
#include "node/subscriber.hpp"
#include "rand_init.hpp"

/**
 *  Ping Pong Test:
 *   1. randomly generate two integers a and b
 *   2. ping side publishes a randomly generated vector v to the pong side
 *   3. pong side publishes back the result of a*v + b
 *   4. ping side also calculates a*v + b and compare with the received one
 */

using namespace std;

const char kTopicNamePing[] = "pp-ping";
const char kTopicNamePong[] = "pp-pong";
const size_t kPoolSize = 2 * 1024 * 1024;  // 2 MiB
const size_t kBufSize = 1024;              // 1 KiB
const size_t kBufCount = kBufSize / sizeof(int);
const int kTotalTimes = 5;

void runAsPingSide(int a, int b);
void runAsPongSide(int a, int b);

int main() {
    srand(time(nullptr));

    int a = rand() % 1000;
    int b = rand() % 1000;

    switch (fork()) {
    case -1:
        cerr << "Failed to fork" << endl;
        return 1;
    case 0:
        runAsPongSide(a, b);
        return 0;
    default:
        runAsPingSide(a, b);
    }

    return 0;
}

__global__ void __vecTransform(int* vec_o, int* vec_i, int a, int b) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= kBufCount) return;
    vec_o[idx] = a * vec_i[idx] + b;
}

void vecTransform(int* vec_o, int* vec_i, int a, int b) {
    int grid_dim = 1;
    int block_dim = kBufCount;
    if (kBufCount > 1024) {
        grid_dim = ((kBufCount - 1) >> 10) + 1;
        block_dim = 1024;
    }
    __vecTransform<<<grid_dim, block_dim>>>(vec_o, vec_i, a, b);
    hipDeviceSynchronize();
}

void runAsPingSide(int a, int b) {
    // init random state
    hiprandState* states;
    hipMalloc(&states, sizeof(hiprandState) * kBufCount);
    initRandStates(states, kBufCount, time(nullptr));

    int* res_d;
    hipMalloc(&res_d, kBufSize);

    int* res = (int*) std::malloc(kBufSize);
    int* data = (int*) std::malloc(kBufSize);

    try {
        mutex cv_m;
        condition_variable cv;

        char runtime_name[32];
        sprintf(runtime_name, "ping-pong-publisher");
        iox::runtime::PoshRuntime::initRuntime(runtime_name);
        Publisher publisher(kTopicNamePing, kPoolSize);

        auto handler = [data, res, &cv](void* data_d, size_t size) {
            hipMemcpy(data, data_d, kBufSize, hipMemcpyDeviceToHost);
            if (memcmp(res, data, kBufSize) == 0) {
                cout << "Passed!" << endl;
            } else {
                cout << "Failed!" << endl;
            }
            cv.notify_one();
        };
        Subscriber subscriber(kTopicNamePong, kPoolSize, handler);

        // make sure both sides are ready
        cout << "Ready..." << endl;
        this_thread::sleep_for(2s);
        cout << "Start!" << endl;

        unique_lock lock{cv_m};
        for (int i = 0; i < kTotalTimes; i++) {
            // generate random vector
            int* vec_d = (int*) publisher.malloc(kBufSize);
            fillRandVals(states, vec_d, kBufCount);
            vecTransform(res_d, vec_d, a, b);
            hipMemcpy(res, res_d, kBufSize, hipMemcpyDeviceToHost);

            // publish the random vector
            publisher.put(vec_d, kBufSize);
            cout << "- - -" << endl;
            cout << "Ping!" << endl;
            cv.wait_for(lock, 1s);
        }
        lock.unlock();
    } catch (runtime_error& err) {
        cerr << "Ping Side: " << err.what() << endl;
        exit(1);
    }

    std::free(res);
    std::free(data);
    hipFree(res_d);
    hipFree(states);
}

void runAsPongSide(int a, int b) {
    try {
        mutex cv_m;
        condition_variable cv;
        int times = 0;

        char runtime_name[32];
        sprintf(runtime_name, "ping-pong-subscriber");
        iox::runtime::PoshRuntime::initRuntime(runtime_name);
        Publisher publisher(kTopicNamePong, kPoolSize);
        int* res_d = (int*) publisher.malloc(kBufSize);

        auto handler = [&publisher, res_d, a, b, &cv, &times](void* data,
                                                              size_t size) {
            vecTransform(res_d, (int*) data, a, b);
            publisher.put(res_d, kBufSize);
            cout << "Pong!" << endl;
            times++;
            cv.notify_one();
        };
        Subscriber subscriber(kTopicNamePing, kPoolSize, handler);

        unique_lock lock{cv_m};
        cv.wait(lock, [&times] { return times == kTotalTimes; });
        lock.unlock();
    } catch (runtime_error& err) {
        cerr << "Pong Side: " << err.what() << endl;
        exit(1);
    }
}