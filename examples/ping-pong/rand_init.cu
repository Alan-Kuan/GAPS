#include "hip/hip_runtime.h"
#include "rand_init.hpp"

#include <hiprand/hiprand_kernel.h>

__global__ void __initRandStates(hiprandState* states, unsigned long seed) {
    size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
    hiprand_init(seed, tid, 0, states + tid);
}

__global__ void __fillRandVals(hiprandState* states, int* arr) {
    size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
    arr[tid] = hiprand_uniform(states + tid) * 10;
}

void initRandStates(hiprandState* states, size_t count, unsigned long seed) {
    dim3 grid_dim = 1;
    dim3 block_dim = count;
    if (count > 1024) {
        grid_dim = (count - 1) / 1024 + 1;
        block_dim = 1024;
    }
    __initRandStates<<<grid_dim, block_dim>>>(states, seed);
    hipDeviceSynchronize();
}

void fillRandVals(hiprandState* states, int* arr, size_t count) {
    dim3 grid_dim = 1;
    dim3 block_dim = count;
    if (count > 1024) {
        grid_dim = (count - 1) / 1024 + 1;
        block_dim = 1024;
    }
    __fillRandVals<<<grid_dim, block_dim>>>(states, arr);
    hipDeviceSynchronize();
}