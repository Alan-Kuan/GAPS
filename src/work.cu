#include "hip/hip_runtime.h"
#include "work.hpp"

#include <cstddef>
#include <iostream>

__global__ void vecMul(int* arr, int mul) {
    arr[threadIdx.x] *= mul;
}

void initAndCopyDataToHost(int* buf, int count) {
    int* arr;
    int* arr_dev;
    std::size_t size = sizeof(int) * count;

    arr = new int[count];
    hipMalloc(&arr_dev, size);

    for (int i = 0; i < count; i++) arr[i] = i;
    hipMemcpy(arr_dev, arr, size, hipMemcpyHostToDevice);
    hipMemcpy(buf, arr_dev, size, hipMemcpyDeviceToHost);

    hipFree(arr_dev);
    delete[] arr;
}

void copyDataToDeviceAndRun(int* buf, int count) {
    int* arr;
    int* arr_dev;
    std::size_t size = sizeof(int) * count;

    arr = new int[count];
    hipMalloc(&arr_dev, size);

    hipMemcpy(arr_dev, buf, size, hipMemcpyHostToDevice);
    vecMul<<<1, count>>>(arr_dev, 2);
    hipMemcpy(arr, arr_dev, size, hipMemcpyDeviceToHost);

    for (int i = 0; i < count; i++) {
        std::cout << arr[i] << ' ';
    }
    std::cout << std::endl;

    hipFree(arr_dev);
    delete[] arr;
}