#include "allocator/shareable.hpp"

#include <sys/stat.h>
#include <sys/socket.h>
#include <sys/un.h>
#include <unistd.h>

#include <cstddef>
#include <cstdio>
#include <cstring>
#include <source_location>

#include <hip/hip_runtime.h>

#include "error.hpp"

namespace {
void throwOnErrorCuda(hipError_t res, std::source_location loc = std::source_location::current()) {
    if (res != hipSuccess) {
        const char* msg;
        hipDrvGetErrorString(res, &msg);
        throwError(msg, loc);
    }
}
}

ShareableAllocator::ShareableAllocator(const char* topic_name, size_t pool_size) {
    if (strlen(topic_name) > sizeof(Metadata().topic_name)) throwError();
    this->attachShm(topic_name, sizeof(Metadata));
    this->createPool(pool_size);
    this->attachPool();
    strcpy(this->getMetadata()->topic_name, topic_name);
}

ShareableAllocator::ShareableAllocator(const char* topic_name) {
    this->attachShm(topic_name, sizeof(Metadata));
}

ShareableAllocator::~ShareableAllocator(void) {
    this->detachPool();
    this->detachShm(this->getMetadata()->topic_name, sizeof(Metadata));
}

void ShareableAllocator::createPool(size_t size) {
    hipMemAllocationProp prop = {};
    prop.requestedHandleTypes = hipMemHandleTypePosixFileDescriptor;
    prop.type = hipMemAllocationTypePinned;
    prop.location.type = hipMemLocationTypeDevice;
    prop.location.id = 0;

    size_t padded_size = this->getPaddedSize(size, &prop);
    throwOnErrorCuda(hipMemCreate(&this->handle, padded_size, &prop, 0));
    this->getMetadata()->pool_size = padded_size;
}

void ShareableAllocator::attachPool(void) {
    hipDeviceptr_t dptr;
    hipMemAccessDesc acc_desc;
    size_t& size = this->getMetadata()->pool_size;

    throwOnErrorCuda(hipMemAddressReserve(&dptr, size, 0, 0, 0));
    throwOnErrorCuda(hipMemMap(dptr, size, 0, this->handle, 0));

    acc_desc.location.id = 0;
    acc_desc.location.type = hipMemLocationTypeDevice;
    acc_desc.flags = hipMemAccessFlagsProtReadWrite;
    throwOnErrorCuda(hipMemSetAccess(dptr, size, &acc_desc, 1));

    this->pool_base = (void*) dptr;
}

void ShareableAllocator::detachPool(void) {
    size_t& size = this->getMetadata()->pool_size;
    throwOnErrorCuda(hipMemRelease(this->handle));
    throwOnErrorCuda(hipMemUnmap((hipDeviceptr_t) this->pool_base, size));
    throwOnErrorCuda(hipMemAddressFree((hipDeviceptr_t) this->pool_base, size));
}

size_t ShareableAllocator::getPaddedSize(const size_t size, const hipMemAllocationProp* prop) const {
    size_t gran = 0;
    throwOnErrorCuda(hipMemGetAllocationGranularity(&gran, prop, hipMemAllocationGranularityMinimum));
    return ((size - 1) / gran + 1) * gran;
}

inline ShareableAllocator::Metadata* ShareableAllocator::getMetadata(void) const {
    return (Metadata*) this->shm_base;
}

void ShareableAllocator::shareHandle(int count) {
    // export the handle to a shareable handle
    ShareableHandle sh_handle;
    throwOnErrorCuda(hipMemExportToShareableHandle((void*) &sh_handle,
        this->handle, hipMemHandleTypePosixFileDescriptor, 0));

    // setup a UNIX Domain Socket server
    int sockfd = throwOnError(socket(AF_UNIX, SOCK_STREAM, 0));

    struct sockaddr_un addr;
    memset(&addr, 0, sizeof(addr));
    addr.sun_family = AF_UNIX;
    throwOnError(sprintf(addr.sun_path, "/tmp/shoz/%s-server.sock",
        this->getMetadata()->topic_name));

    throwOnError(bind(sockfd, (struct sockaddr*) &addr, sizeof(addr)));
    throwOnError(listen(sockfd, 8));

    // prepare message
    struct msghdr msg;
    struct iovec iov[1];
    char ctrl_buf[CMSG_SPACE(sizeof(int))];

    // dummy data
    iov[0].iov_base = (void*) "";
    iov[0].iov_len = 1;

    msg.msg_name = nullptr;
    msg.msg_namelen = 0;
    msg.msg_iov = iov;
    msg.msg_iovlen = 1;
    msg.msg_control = ctrl_buf;
    msg.msg_controllen = sizeof(ctrl_buf);

    struct cmsghdr* cmsg = CMSG_FIRSTHDR(&msg);

    cmsg->cmsg_len = CMSG_LEN(sizeof(int));
    cmsg->cmsg_level = SOL_SOCKET;
    cmsg->cmsg_type = SCM_RIGHTS;
    *((int*) CMSG_DATA(cmsg)) = (int) sh_handle;

    // send shareable handle to `count` clients
    for (int N = count; N > 0; N--) {
        int cli_fd = throwOnError(accept(sockfd, nullptr, nullptr));
        throwOnError(sendmsg(cli_fd, &msg, 0));
        throwOnError(close(cli_fd));
    }

    throwOnError(close(sockfd));
    throwOnError(unlink(addr.sun_path));
}

void ShareableAllocator::recvHandle(void) {
    // setup UNIX Domain Socket client
    int sockfd = throwOnError(socket(AF_UNIX, SOCK_STREAM, 0));

    struct sockaddr_un cli_addr;
    memset(&cli_addr, 0, sizeof(cli_addr));
    cli_addr.sun_family = AF_UNIX;
    throwOnError(sprintf(cli_addr.sun_path, "/tmp/shoz/%s-client-%d-%d.sock",
        this->getMetadata()->topic_name, getpid(), gettid()));

    throwOnError(bind(sockfd, (struct sockaddr*) &cli_addr, sizeof(cli_addr)));

    struct sockaddr_un server_addr;
    memset(&server_addr, 0, sizeof(server_addr));
    server_addr.sun_family = AF_UNIX;
    throwOnError(sprintf(server_addr.sun_path, "/tmp/shoz/%s-server.sock",
        this->getMetadata()->topic_name));

    // should wait until the server is ready
    struct stat buf;
    for (int tries = 0; tries < 2 && (stat(server_addr.sun_path, &buf) != 0); tries++) {
        usleep(50000);
    }
    throwOnError(connect(sockfd, (struct sockaddr*) &server_addr, sizeof(server_addr)));

    // prepare message
    struct msghdr msg;
    struct iovec iov[1];
    char ctrl_buf[CMSG_SPACE(sizeof(int))];
    char dummy[1];

    iov[0].iov_base = dummy;
    iov[0].iov_len = 1;

    msg.msg_name = nullptr;
    msg.msg_namelen = 0;
    msg.msg_iov = iov;
    msg.msg_iovlen = 1;
    msg.msg_control = ctrl_buf;
    msg.msg_controllen = sizeof(ctrl_buf);

    // receive the message with the shareable handle
    throwOnError(recvmsg(sockfd, &msg, 0));

    throwOnError(close(sockfd));
    throwOnError(unlink(cli_addr.sun_path));

    // import the shareable handle into a generic handle
    struct cmsghdr* cmsg = CMSG_FIRSTHDR(&msg);
    if (!cmsg || cmsg->cmsg_len != CMSG_LEN(sizeof(int))) throwError();
    
    ShareableHandle sh_handle = *((ShareableHandle*) CMSG_DATA(cmsg));
    throwOnErrorCuda(hipMemImportFromShareableHandle(&this->handle,
        (void*) (uintptr_t) sh_handle, hipMemHandleTypePosixFileDescriptor));

    this->attachPool();
}

// TODO: implement TLSF strategy
void* ShareableAllocator::malloc(size_t size) {
    return this->pool_base;
}

// TODO: implement TLSF strategy
void ShareableAllocator::free(void* ptr) {

}