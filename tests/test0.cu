#include "hip/hip_runtime.h"
#include <fcntl.h>
#include <semaphore.h>
#include <sys/time.h>
#include <unistd.h>

#include <chrono>
#include <cstddef>
#include <cstdlib>
#include <ctime>
#include <iostream>
#include <stdexcept>

#include <hip/hip_runtime.h>
#include <zenoh.hxx>

#include "allocator/allocator.hpp"
#include "helpers.hpp"
#include "node/publisher.hpp"
#include "node/subscriber.hpp"

using namespace std;
using namespace hlp;

__global__ void __vecAdd(int* c, int* a, int* b) {
    c[threadIdx.x] = a[threadIdx.x] + b[threadIdx.x];
}

/* Global */
#define POOL_SIZE 65536
const char kDftLLocator[] = "udp/224.0.0.123:7447#iface=lo";
Domain domain = {DeviceType::kGPU, 0};
sem_t* sem;
size_t transmit_size = 0;
size_t asize = 0;

void pubTest(const char* zenConfig) {
    Timer timer;
    try {
        hipInit(0);
        timer.setPoint();
        Publisher pub("topic 0", zenConfig, domain, POOL_SIZE);
        timer.setPoint();

        int* arr = new int[asize];

        for (int i = 0; i < asize; i++) arr[i] = rand() % 10;

        sem_wait(sem);
        timer.setPoint();
        // pub.put(arr, sizeof(int) * 1024);
        pub.put(arr, transmit_size);
        timer.setPoint();

        delete[] arr;

    } catch (zenoh::ErrorMessage& err) {
        cerr << "Zenoh: " << err.as_string_view() << endl;
        exit(1);
    } catch (runtime_error& err) {
        cerr << "Publisher: " << err.what() << endl;
        exit(1);
    }
    timer.showAll("pub");
}

void subTest(const char* zenConfig) {
    Timer timer;
    try {
        hipInit(0);
        Subscriber sub("topic 0", zenConfig, domain, POOL_SIZE);
        Subscriber::MessageHandler handler;

        bool handleEnd = false;

        int* c;
        size_t vsize = asize / 2;  // decompose data from publisher
        hipMalloc(&c, sizeof(int) * vsize);

        handler = [c, vsize, &handleEnd, &timer](void* msg, size_t size) {
            // std::cout << "Is it all ready?: " << size << std::endl;

            timer.setPoint();
            int* arr = new int[vsize];
            int* a = (int*) msg;
            int* b = (int*) msg + vsize;

            timer.setPoint();
            __vecAdd<<<1, 512>>>(c, a, b);
            timer.setPoint();

            timer.setPoint();
            hipMemcpy(arr, c, sizeof(int) * vsize, hipMemcpyDeviceToHost);
            timer.setPoint();
            // cout << "a + b:" << endl;
            // for (int i = 0; i < 512; i++) cout << arr[i] << ' ';
            // cout << endl;
            handleEnd = true;
        };

        timer.setPoint();
        sub.sub(handler);
        timer.setPoint();
        sem_post(sem);

        std::cout << handleEnd << "busy waiting\n";
        while (!handleEnd) {
        }
        std::cout << handleEnd << "leave\n";
    } catch (zenoh::ErrorMessage& err) {
        cerr << "Zenoh: " << err.as_string_view() << endl;
        exit(1);
    } catch (runtime_error& err) {
        cerr << "Subscriber: " << err.what() << endl;
        exit(1);
    }

    timer.showAll("sub");
}

int main(int argc, char* argv[]) {
    if (argc < 2) {
        cerr << "usage: ./test0 [size](4~" << POOL_SIZE << ")\n";
        exit(1);
    }

    const char* config = kDftLLocator;

    sem = sem_open("/sem_share", O_CREAT, 0660, 0);
    transmit_size = stoul(argv[1]);
    asize = transmit_size / sizeof(int);

    switch (fork()) {
    case -1:
        return -1;
    case 0:
        pubTest(config);
        break;
    default:
        subTest(config);
    }

    sem_close(sem);
    sem_unlink("/sem_share");
    return 0;
}
