#include "hip/hip_runtime.h"
// #define ZENOHCXX_ZENOHC

#include <sys/time.h>
#include <unistd.h>

#include <chrono>
#include <cstddef>
#include <cstdlib>
#include <ctime>
#include <iostream>
#include <stdexcept>

#include <hip/hip_runtime.h>
#include <zenoh.hxx>

#include "allocator/allocator.hpp"
#include "examples/vector_arithmetic.hpp"
#include "helpers.hpp"
#include "node/publisher.hpp"
#include "node/subscriber.hpp"

using namespace std;
using namespace hlp;

__global__ void __vecAdd(int* c, int* a, int* b) {
    c[threadIdx.x] = a[threadIdx.x] + b[threadIdx.x];
}

/* Global */
TimePoint beginTime;
TimePoint endTime;
Domain domain = {DeviceType::kGPU, 0};

void pubTest(char* config_path) {
    try {
        hipInit(0);
        Publisher pub("topic 0", config_path, domain, 4096);

        int arr[1024];

        for (int i = 0; i < 1024; i++) arr[i] = rand() % 10;

        beginTime.set();
        pub.put(arr, sizeof(int) * 1024);
        endTime.set();

        sleep(1);

        cout << "pub begin: " << fixed << beginTime.getMSec();
        cout << ", end: " << fixed << endTime.getMSec() << endl;

    } catch (zenoh::ErrorMessage& err) {
        cerr << "Zenoh: " << err.as_string_view() << endl;
        exit(1);
    } catch (runtime_error& err) {
        cerr << "Publisher: " << err.what() << endl;
        exit(1);
    }
}

void subTest(char* config_path) {
    try {
        hipInit(0);
        Subscriber sub("topic 0", config_path, domain, 4096);
        Subscriber::MessageHandler handler;

        int* c;
        hipMalloc(&c, sizeof(int) * 512);

        handler = [c](void* msg, size_t size) {
            beginTime.set();
            int arr[512];
            int* a = (int*) msg;
            int* b = (int*) msg + 512;

            __vecAdd<<<1, 512>>>(c, a, b);

            hipMemcpy(arr, c, sizeof(int) * 512, hipMemcpyDeviceToHost);
            // cout << "a + b:" << endl;
            // for (int i = 0; i < 512; i++) cout << arr[i] << ' ';
            // cout << endl;

            cout << "sub handle begin: " << fixed << beginTime.getMSec()
                 << endl;
        };

        sub.sub(handler);
        sleep(5);
    } catch (zenoh::ErrorMessage& err) {
        cerr << "Zenoh: " << err.as_string_view() << endl;
        exit(1);
    } catch (runtime_error& err) {
        cerr << "Subscriber: " << err.what() << endl;
        exit(1);
    }
}

int main(int argc, char* argv[]) {
    char* config_path = argv[1];

    switch (fork()) {
    case -1:
        return -1;
    case 0:
        pubTest(config_path);
        break;
    default:
        subTest(config_path);
    }

    return 0;
}
