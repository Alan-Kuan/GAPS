#include "hip/hip_runtime.h"
#include <fcntl.h>
#include <semaphore.h>
#include <sys/time.h>
#include <unistd.h>

#include <chrono>
#include <cstddef>
#include <cstdlib>
#include <ctime>
#include <iostream>
#include <stdexcept>

#include <hip/hip_runtime.h>
#include <zenoh.hxx>

#include "allocator/allocator.hpp"
#include "helpers.hpp"
#include "node/publisher.hpp"
#include "node/subscriber.hpp"

using namespace std;
using namespace hlp;

__global__ void __vecAdd(int* c, int* a, int* b) {
    c[threadIdx.x] = a[threadIdx.x] + b[threadIdx.x];
}

/* Global */
#define POOL_SIZE 4096
const char kDftLLocator[] = "udp/224.0.0.123:7447#iface=lo";
Timer timer;
Domain domain = {DeviceType::kGPU, 0};
sem_t* sem;
size_t transmit_size = 0;
size_t asize = 0;

void pubTest(const char* zenConfig) {
    try {
        hipInit(0);
        timer.setPoint();
        Publisher pub("topic 0", zenConfig, domain, POOL_SIZE);
        timer.setPoint();

        int* arr = new int[asize];

        for (int i = 0; i < asize; i++) arr[i] = rand() % 10;

        sem_wait(sem);
        timer.setPoint();
        // pub.put(arr, sizeof(int) * 1024);
        pub.put(arr, transmit_size);
        timer.setPoint();

        delete[] arr;

    } catch (zenoh::ErrorMessage& err) {
        cerr << "Zenoh: " << err.as_string_view() << endl;
        exit(1);
    } catch (runtime_error& err) {
        cerr << "Publisher: " << err.what() << endl;
        exit(1);
    }
}

void subTest(const char* zenConfig) {
    try {
        hipInit(0);
        Subscriber sub("topic 0", zenConfig, domain, POOL_SIZE);
        Subscriber::MessageHandler handler;

        int* c;
        size_t vsize = asize / 2;  // decompose data from publisher
        hipMalloc(&c, sizeof(int) * vsize);

        handler = [c, vsize](void* msg, size_t size) {
            // std::cout << "Is it all ready?: " << size << std::endl;

            timer.setPoint();
            int* arr = new int[vsize];
            int* a = (int*) msg;
            int* b = (int*) msg + vsize;

            timer.setPoint();
            __vecAdd<<<1, 512>>>(c, a, b);
            timer.setPoint();

            timer.setPoint();
            hipMemcpy(arr, c, sizeof(int) * vsize, hipMemcpyDeviceToHost);
            timer.setPoint();
            // cout << "a + b:" << endl;
            // for (int i = 0; i < 512; i++) cout << arr[i] << ' ';
            // cout << endl;
        };

        timer.setPoint();
        sub.sub(handler);
        timer.setPoint();
        std::cout << "regist done, ready to post\n";
        sem_post(sem);

        sleep(1);  // waiting publisher to transmit data
    } catch (zenoh::ErrorMessage& err) {
        cerr << "Zenoh: " << err.as_string_view() << endl;
        exit(1);
    } catch (runtime_error& err) {
        cerr << "Subscriber: " << err.what() << endl;
        exit(1);
    }
}

int main(int argc, char* argv[]) {
    const char* config = kDftLLocator;

    sem = sem_open("/sem_share", O_CREAT, 0660, 0);
    transmit_size = stoul(argv[1]);
    asize = transmit_size / sizeof(int);

    switch (fork()) {
    case -1:
        return -1;
    case 0:
        pubTest(config);
        cout << "pub:\n";
        timer.showAll();
        break;
    default:
        subTest(config);
        cout << "sub:\n";
        timer.showAll();
    }

    sem_close(sem);
    sem_unlink("/sem_share");
    return 0;
}
